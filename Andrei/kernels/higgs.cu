#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include "grids.h"

template<int group_size, int codebook_bits, int scales_size>
__global__ void HiggsAlignedMatVec(
  const uint4* __restrict__ A,
  const uint4* __restrict__ B,
        uint4* __restrict__ C,
  const half* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  constexpr int codes_in_half = 16 / codebook_bits;
  constexpr int halfs_in_uint4 = 128 / 16;
  constexpr int threads_in_wave = 32;
  constexpr int steps_in_wave = scales_size / (threads_in_wave * 8);

  int a_gl_stride_half = prob_k / group_size / codes_in_half;
  int c_gl_wr = (blockDim.x / threads_in_wave) * blockIdx.x + (threadIdx.x / threads_in_wave);
  bool pred = c_gl_wr < prob_m;
  int b_gl_rd = 0;
  int a_gl_rd_half = a_gl_stride_half * c_gl_wr + (threadIdx.x % threads_in_wave) * (scales_size / group_size) / threads_in_wave / codes_in_half;
  int a_gl_end_half = a_gl_stride_half * (c_gl_wr + 1);

  constexpr int replication = 1024 / 16 / group_size;
  int lane = threadIdx.x % replication;
  __shared__ uint4 sh_grid[replication * 256 * group_size / halfs_in_uint4];
  const uint4* codebook;
  if constexpr (group_size == 2 && codebook_bits == 8) {
    codebook = HIGGS_2_256;
  } else if constexpr (group_size == 4 && codebook_bits == 8) {
    codebook = HIGGS_4_256;
  } else if constexpr (group_size == 1 && codebook_bits == 8) {
    codebook = HIGGS_1_256;
  }
  
  for (int i = threadIdx.x; i < 256 * group_size / halfs_in_uint4; i += blockDim.x) {
    uint4 dec = codebook[i];
    #pragma unroll
    for (int j = 0; j < replication; j++)
      sh_grid[replication * i + (j + lane) % replication] = dec;
  }
  __syncthreads();

  __shared__ uint4 sh_b[threads_in_wave * (steps_in_wave + 1)];
  float res = 0;

  int iters = (prob_k - 1) / scales_size + 1;
  while (iters--) {
    // We pad shared memory to avoid bank conflicts during reads
    __syncthreads();
    for (int i = threadIdx.x; i < threads_in_wave * steps_in_wave; i += blockDim.x) {
      if (steps_in_wave * (b_gl_rd + i) < prob_k)
        sh_b[(steps_in_wave + 1) * (i / steps_in_wave) + i % steps_in_wave] = B[b_gl_rd + i];
    }
    __syncthreads();

    float iter_res = 0;

    int b_sh_rd = (steps_in_wave + 1) * (threadIdx.x % threads_in_wave);
    if (pred && a_gl_rd_half < a_gl_end_half) {
      float scale = __half2float(scales[a_gl_rd_half * codes_in_half * group_size / scales_size]);
      const uint8_t* enc = reinterpret_cast<const uint8_t*>(reinterpret_cast<const half*>(A) + a_gl_rd_half);
            
      #pragma unroll
      for (int i = 0; i < steps_in_wave; i++) {
        uint32_t dec[4];
        #pragma unroll
        for (int j = 0; j < (8 - 1) / group_size + 1; j++) {
          if constexpr (group_size == 2 && codebook_bits == 8) {
            ((uint32_t*)dec)[j] = *(((const uint32_t*)sh_grid) + replication * enc[(8 / group_size) * i + j] + lane); // read 2 halfs at a time
          } else if constexpr (group_size == 4 && codebook_bits == 8) {
            ((uint64_t*)dec)[j] = *(((const uint64_t*)sh_grid) + replication * enc[(8 / group_size) * i + j] + lane); // read 4 halfs at a time
          } else if constexpr (group_size == 1 && codebook_bits == 8) {
            ((uint16_t*)dec)[j] = *(((const uint16_t*)sh_grid) + replication * enc[(8 / group_size) * i + j] + lane); // read 1 halfs at a time
          }
        }
        
        half2* a = reinterpret_cast<half2*>(&dec);
        half2* b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
        half2 res2 = {};
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          res2 = __hfma2(a[j], b[j], res2);
        }
        iter_res += __half2float(res2.x) + __half2float(res2.y);
        b_sh_rd += 1;
      }
      iter_res *= scale;
      a_gl_rd_half += scales_size / group_size / codes_in_half;
    }
    b_gl_rd += threads_in_wave * steps_in_wave; // Move by scales_size
    res += iter_res;
  }

  if (pred) {
    #pragma unroll
    for (int i = 16; i > 0; i /= 2)
      res += __shfl_down_sync(0xffffffff, res, i);
    if (threadIdx.x % threads_in_wave == 0) {
      reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
    }
  }
}

inline int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

const int THREAD_M = 16;


template<int group_size, int codebook_bits, int scales_size>
void  higgs_aligned_matvec_cuda(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  int cc_major;
  hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, 0);

  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  int shared = 16 * 32 * (scales_size / 32 / 8 + 1) + 16 * group_size * (1 << codebook_bits);
  hipFuncSetAttribute(reinterpret_cast<const void*>(
    HiggsAlignedMatVec<group_size), codebook_bits, scales_size>, hipFuncAttributeMaxDynamicSharedMemorySize, shared
  );
  HiggsAlignedMatVec<group_size, codebook_bits, scales_size><<<blocks, threads, shared>>>(
    (const uint4*) A,
    (const uint4*) B,
    (uint4*) C,
    (const half*) scales,
    prob_m,
    prob_k
  );
}

template void  higgs_aligned_matvec_cuda<2, 8, 1024>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);

template void  higgs_aligned_matvec_cuda<4, 8, 1024>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);

template void  higgs_aligned_matvec_cuda<1, 8, 1024>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);

template<int group_size>
__global__ void HiggsKx256MatVec(
  const uint4* __restrict__ A,
  const uint4* __restrict__ B,
        uint4* __restrict__ C,
  const half* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  constexpr int scales_size = 1024;
  constexpr int codebook_bits = 8;
  constexpr int steps_in_wave = 4;
  constexpr int threads_in_wave = 32;
  const half* grid;
  if constexpr (group_size == 3) {
    grid = (const half*)HIGGS_3_256;
  } else if constexpr (group_size == 5) {
    grid = (const half*)HIGGS_5_256;
  } else if constexpr (group_size == 6) {
    grid = (const half*)HIGGS_6_256;
  }
  

  const int a_gl_stride_8 =  (prob_k / scales_size) * ((scales_size - 1) / group_size + 1);
  const int c_gl_wr = (blockDim.x / threads_in_wave) * blockIdx.x + (threadIdx.x / threads_in_wave);
  const bool pred = c_gl_wr < prob_m;
  int b_gl_rd = 0;
  int a_gl_rd_8 = a_gl_stride_8 * c_gl_wr + (threadIdx.x % threads_in_wave) * (scales_size / threads_in_wave) / group_size;
  const int zeroth_offset = (((threadIdx.x % threads_in_wave) % group_size) * 2) % group_size;

  int a_gl_end_8 = a_gl_stride_8 * (c_gl_wr + 1);

  __shared__ uint4 sh_b[threads_in_wave * (steps_in_wave + 1)];
  float res = 0;
  
  int iters = (prob_k - 1) / scales_size + 1;
  while (iters--) {
    // We pad shared memory to avoid bank conflicts during reads
    __syncthreads();
    for (int i = threadIdx.x; i < threads_in_wave * steps_in_wave; i += blockDim.x) {
      if (steps_in_wave * (b_gl_rd + i) < prob_k)
        sh_b[(steps_in_wave + 1) * (i / steps_in_wave) + i % steps_in_wave] = B[b_gl_rd + i];
    }
    __syncthreads();

    float iter_res = 0;

    int b_sh_rd = (steps_in_wave + 1) * (threadIdx.x % threads_in_wave);
    if (pred && a_gl_rd_8 < a_gl_end_8) {
      float scale = __half2float(scales[a_gl_rd_8 / ((scales_size - 1) / group_size + 1)]);
      const uint8_t* enc = reinterpret_cast<const uint8_t*>(A) + a_gl_rd_8;

      constexpr int num_values_to_load = ((32 - 1) / group_size + 2) * group_size;
      half dec[num_values_to_load];
      #pragma unroll
      for (int i = 0; i < num_values_to_load / group_size; i++) {
        #pragma unroll
        for (int j = 0; j < group_size; j++) {
          dec[group_size * i + j] = __ldca((grid + group_size * enc[i] + j)); // read 1 halfs at a time
        }
      }

      half* a = reinterpret_cast<half*>(&dec[zeroth_offset]);
      half* b = reinterpret_cast<half*>(&sh_b[b_sh_rd]);
      
      #pragma unroll
      for (int i = 0; i < 4; i++) {
        half res_half = {};
        #pragma unroll
        for (int j = 0; j < 8; j++) {
          res_half = __hfma(a[8 * i + j], b[8 * i + j], res_half);
        }
        iter_res += __half2float(res_half);
      }
      
      iter_res *= scale;
      a_gl_rd_8 += (scales_size - 1) / group_size + 1;
    }
    b_gl_rd += threads_in_wave * steps_in_wave; // Move by scales_size
    res += iter_res;
  }

  if (pred) {
    #pragma unroll
    for (int i = 16; i > 0; i /= 2)
      res += __shfl_down_sync(0xffffffff, res, i);
    if (threadIdx.x % threads_in_wave == 0) {
      reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
    }
  }
}

template<int group_size>
void  higgs_Kx256_matvec_cuda(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  int cc_major;
  hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, 0);

  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  constexpr int scales_size = 1024;
  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  int shared = 16 * 32 * (scales_size / 32 / 8 + 1);
  hipFuncSetAttribute(reinterpret_cast<const void*>(
    HiggsKx256MatVec<group_size>), hipFuncAttributeMaxDynamicSharedMemorySize, shared
  );
  HiggsKx256MatVec<group_size><<<blocks, threads, shared>>>(
    (const uint4*) A,
    (const uint4*) B,
    (uint4*) C,
    (const half*) scales,
    prob_m,
    prob_k
  );
}

template void  higgs_Kx256_matvec_cuda<3>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);

template void  higgs_Kx256_matvec_cuda<5>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);

template void  higgs_Kx256_matvec_cuda<6>(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
);
