#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include "grids.h"

template<int group_size=2, int codebook_bits=8, int hadamard_size=1024>
__global__ void Higgs2x256MatVec(
  const int4* __restrict__ A,
  const int4* __restrict__ B,
        int4* __restrict__ C,
  const half* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  constexpr int halfs_in_int4 = 128 / 16;
  constexpr int steps_in_wave = hadamard_size / 256;
  constexpr int threads_in_wave = 32;

  int a_gl_stride = prob_k / halfs_in_int4 / steps_in_wave;
  int a_gl_rd = (blockDim.x / threads_in_wave) * blockIdx.x + (threadIdx.x / threads_in_wave);
  int row_number = a_gl_rd;
  bool pred = a_gl_rd < prob_m;
  int b_gl_rd = 0;
  int c_gl_wr = a_gl_rd;
  a_gl_rd = a_gl_stride * a_gl_rd + threadIdx.x % threads_in_wave;
  int a_gl_end = a_gl_rd + a_gl_stride - threadIdx.x % threads_in_wave;

  __shared__ int4 sh_b[threads_in_wave * (steps_in_wave + 1)];
  float res = 0;

  int iters = (prob_k - 1) / hadamard_size + 1;
  while (iters--) {
    // We pad shared memory to avoid bank conflicts during reads
    __syncthreads();
    for (int i = threadIdx.x; i < threads_in_wave * steps_in_wave; i += blockDim.x) {
      if (steps_in_wave * (b_gl_rd + i) < prob_k)
        sh_b[(steps_in_wave + 1) * (i / steps_in_wave) + i % steps_in_wave] = B[b_gl_rd + i];
    }
    __syncthreads();

    float iter_res = 0;

    int b_sh_rd = (steps_in_wave + 1) * (threadIdx.x % threads_in_wave);
    if (pred && a_gl_rd < a_gl_end) {
      float scale = __half2float(scales[(a_gl_rd * (128 / codebook_bits) * group_size) / hadamard_size]);
      const uint8_t* enc = reinterpret_cast<const uint8_t*>(&A[a_gl_rd]);
            
      #pragma unroll
      for (int i = 0; i < steps_in_wave; i++) {
        uint32_t dec[4];
        #pragma unroll
        for (int j = 0; j < 8 / group_size; j++) {
          if constexpr (group_size == 2 && codebook_bits == 8) {
            ((uint32_t*)dec)[j] = ((uint32_t*)HIGGS_2_256)[enc[(8 / group_size) * i + j]]; // read 2 halfs at a time
          } else if constexpr (group_size == 2 && codebook_bits == 8) {
            ((uint64_t*)dec)[j] = ((uint64_t*)HIGGS_4_256)[enc[(8 / group_size) * i + j]]; // read 4 halfs at a time
          }
        }
        
        half2* a = reinterpret_cast<half2*>(&dec);
        half2* b = reinterpret_cast<half2*>(&sh_b[b_sh_rd]);
        half2 res2 = {};
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          res2 = __hfma2(a[j], b[j], res2);
        }
        iter_res += __half2float(res2.x) + __half2float(res2.y);
        b_sh_rd += 1;
      }
      iter_res *= scale;
      a_gl_rd += threads_in_wave;
    }
    b_gl_rd += threads_in_wave * steps_in_wave; // Move by hadamard_size
    res += iter_res;
  }

  if (pred) {
    #pragma unroll
    for (int i = 16; i > 0; i /= 2)
      res += __shfl_down_sync(0xffffffff, res, i);
    if (threadIdx.x % threads_in_wave == 0) {
      reinterpret_cast<__half*>(C)[c_gl_wr] = __float2half(res);
    }
  }
}

inline int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

const int THREAD_M = 16;

void  higgs2x256_matvec_cuda(
  const void* __restrict__ A,
  const void* __restrict__ B,
        void* __restrict__ C,
  const void* __restrict__ scales,
  int prob_m,
  int prob_k
) {
  int cc_major;
  hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, 0);

  int sms;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
  int waves = 0;
  int thread_m;
  do {
    waves++;
    thread_m = ceildiv(prob_m, waves * sms);
  } while (thread_m > THREAD_M);

  int blocks = ceildiv(prob_m, thread_m);
  int threads = 32 * thread_m;
  int shared = 16 * 32 * 9;
  hipFuncSetAttribute(reinterpret_cast<const void*>(
    Higgs2x256MatVec<>), hipFuncAttributeMaxDynamicSharedMemorySize, shared
  );
  Higgs2x256MatVec<><<<blocks, threads, shared>>>(
    (const int4*) A,
    (const int4*) B,
    (int4*) C,
    (const half*) scales,
    prob_m,
    prob_k
  );
}

// #define CUDACHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
// inline void cuda_check(hipError_t error_code, const char *file, int line)
// {
//     if (error_code != hipSuccess)
//     {
//         fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
//         fflush(stderr);
//         exit(error_code);
//     }
// }

// int main() {
//     const auto codes = std::vector<uint8_t>(1024 * 1024 / 2, 1);
//     uint8_t* codes_device;
//     hipMalloc(&codes_device, 1024 * 1024 / 2);
//     hipMemcpy((void**)codes_device, codes.data(), 1024 * 1024 / 2, hipMemcpyHostToDevice);
    
//     const auto scales = std::vector<__half>(1024, 0.0001);
//     __half* scales_device;
//     hipMalloc(&scales_device, 1024 * 2);
//     hipMemcpy((void**)scales_device, scales.data(), 1024 * 2, hipMemcpyHostToDevice);

//     const auto input = std::vector<__half>(1024, 1);
//     __half* input_device;
//     hipMalloc(&input_device, 1024 * 2);
//     hipMemcpy((void**)input_device, input.data(), 1024 * 2, hipMemcpyHostToDevice);

//     auto output = std::vector<__half>(1024, 0);
//     __half* output_device;
//     hipMalloc(&output_device, 1024 * 2);
//     hipMemcpy((void**)output_device, output.data(), 1024 * 2, hipMemcpyHostToDevice);

//     higgs2x256_matvec_cuda(
//         codes_device,
//         input_device,
//         output_device,
//         scales_device,
//         1024,
//         1024
//     );
//     hipMemcpy((void**)output.data(), output_device, 1024 * 2, hipMemcpyDeviceToHost);
//     CUDACHECK(hipPeekAtLastError());

//     std::cout << static_cast<float>(output[0]) << std::endl;

//     return 0;
// };
